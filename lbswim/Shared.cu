#include "Shared.h"
#include "cucall.h"

template <typename T>
SharedArray<T>::SharedArray(size_t n) : size(n) {
  host = new T[n];
  CUDA_SAFE_CALL(hipMalloc(&device, n * sizeof(T)));
}
template <typename T>
SharedArray<T>::~SharedArray() {
  delete[] host;
  CUDA_SAFE_CALL(hipFree(device));
}
template <typename T>
void SharedArray<T>::H2D() {
  CUDA_SAFE_CALL(hipMemcpy(device,
			    host,
			    size * sizeof(T),
			    hipMemcpyHostToDevice));
}
template <typename T>
void SharedArray<T>::D2H() {
  CUDA_SAFE_CALL(hipMemcpy(host,
			    device,
			    size * sizeof(T),
			    hipMemcpyDeviceToHost));
}

template <typename T>
SharedItem<T>::SharedItem() {
  host = new T;
  CUDA_SAFE_CALL(hipMalloc(&device, sizeof(T)));
}
template <typename T>
SharedItem<T>::~SharedItem() {
  CUDA_SAFE_CALL(hipFree(device));
}
template <typename T>
void SharedItem<T>::H2D() {
  CUDA_SAFE_CALL(hipMemcpy(device,
			    host,
			    sizeof(T),
			    hipMemcpyHostToDevice));
}
template <typename T>
void SharedItem<T>::D2H() {
  CUDA_SAFE_CALL(hipMemcpy(host,
			    device,
			    sizeof(T),
			    hipMemcpyDeviceToHost));
}
