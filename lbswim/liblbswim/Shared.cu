#include "Shared.h"
#include "cucall.h"

template <typename T>
SharedArray<T>::SharedArray(size_t n) : size(n) {
  host = new T[n];
  CUDA_SAFE_CALL(hipMalloc(&device, n * sizeof(T)));
}
template <typename T>
SharedArray<T>::~SharedArray() {
  delete[] host;
  CUDA_SAFE_CALL(hipFree(device));
}
template <typename T>
void SharedArray<T>::H2D() {
  CUDA_SAFE_CALL(hipMemcpy(device,
			    host,
			    size * sizeof(T),
			    hipMemcpyHostToDevice));
}
template <typename T>
void SharedArray<T>::D2H() {
  CUDA_SAFE_CALL(hipMemcpy(host,
			    device,
			    size * sizeof(T),
			    hipMemcpyDeviceToHost));
}

template <typename T>
SharedItem<T>::SharedItem() {
  host = new T;
  CUDA_SAFE_CALL(hipMalloc(&device, sizeof(T)));
}
template <typename T>
SharedItem<T>::SharedItem(const T* init) {
  host = new T;
  *host = *init;
  CUDA_SAFE_CALL(hipMalloc(&device, sizeof(T)));
  H2D();
}
template <typename T>
SharedItem<T>::~SharedItem() {
  delete host;
  CUDA_SAFE_CALL(hipFree(device));
}
template <typename T>
void SharedItem<T>::H2D() {
  CUDA_SAFE_CALL(hipMemcpy(device,
			    host,
			    sizeof(T),
			    hipMemcpyHostToDevice));
}
template <typename T>
void SharedItem<T>::D2H() {
  CUDA_SAFE_CALL(hipMemcpy(host,
			    device,
			    sizeof(T),
			    hipMemcpyDeviceToHost));
}
